#include "hip/hip_runtime.h"
#include "stdio.h"
#define CUDA_ERR_CHECK(x) \
	do{ hipError_t err = x; \
		if (err != hipSuccess) { \
			fprintf(stderr, "Error \"%s\" at %s:%d \n", \
				hipGetErrorString(err), __FILE__, __LINE__);\
		exit(0);} \
	} while(0)

#define DGX 3
#define DGY 2 
#define DBX 2
#define DBY 2 
#define DBZ 2
 
#define N (DBX*DBY*DBZ*DGX*DGY)

__global__ void gpu_kernel() {
	/*
	int block_idx, grid_dim;
	block_idx = blockIdx.x;	//номер блока по оси x
	grid_dim = gridDim.x;	//общее количество блоков по оси х
 	*/
	printf("\nThread:\n   block_x # %d\t   block_y # %d\t   block_z # %d\n   thread_x # %d\t   thread_y # %d\t   thread_z # %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

int main (void){

	dim3 grid(DGX, DGY);
	dim3 block(DBX,DBY,DBZ);
	
	int dev;
	hipDeviceProp_t prop;
	
	CUDA_ERR_CHECK( hipGetDevice( &dev ) );
	CUDA_ERR_CHECK( hipGetDeviceProperties(&prop, dev) );

	printf("name\t\t\t%s\n", prop.name);
	printf("totalGlobalMem\t\t%zd\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock\t%zd\n", prop.sharedMemPerBlock);
	printf("regsPerBlock\t\t%d\n", prop.regsPerBlock);
	printf("warpSize\t\t%d\n", prop.warpSize);
	printf("memPitch\t\t%zd\n", prop.memPitch);
	printf("maxThreadsPerBlock\t%d\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim.x\t\t%d\n", prop.maxThreadsDim[0]);
	printf("maxThreadsDim.y\t\t%d\n", prop.maxThreadsDim[1]);
	printf("maxThreadsDim.z\t\t%d\n", prop.maxThreadsDim[2]);
	printf("maxGridSize.x\t\t%d\n", prop.maxGridSize[0]);
	printf("maxGridSize.y\t\t%d\n", prop.maxGridSize[1]);
	printf("maxGridSize.z\t\t%d\n", prop.maxGridSize[2]);
	printf("totalConstMem\t\t%zd\n", prop.totalConstMem);
	printf("major\t\t\t%d\n", prop.major);
	printf("minor\t\t\t%d\n", prop.minor);
	printf("clockRate\t\t%d\n", prop.clockRate);
	printf("textureAlignment\t%zd\n", prop.textureAlignment);
	printf("deviceOverlap\t\t%d\n", prop.deviceOverlap);
	printf("multiProcessorCount\t%d\n", prop.multiProcessorCount);
	printf("kernelExecTimeoutEnabled %d\n", prop.kernelExecTimeoutEnabled);
	printf("integrated\t\t%d\n", prop.integrated);
	printf("canMapHostMemory\t%d\n", prop.canMapHostMemory);
	printf("computeMode\t\t%d\n", prop.computeMode);
	printf("concurrentKernels\t%d\n", prop.concurrentKernels);
	printf("ECCEnabled\t\t%d\n", prop.ECCEnabled);
	printf("pciBusID\t\t%d\n", prop.pciBusID);
	printf("pciDeviceID\t\t%d\n", prop.pciDeviceID);
	printf("tccDriver\t\t%d\n", prop.tccDriver);

	printf("hipComputeMode:\n");
	printf("hipComputeModeDefault: %d\n", hipComputeModeDefault);
	printf("hipComputeModeExclusive: %d\n", hipComputeModeExclusive);
	printf("hipComputeModeProhibited: %d\n", hipComputeModeProhibited);	

		
	//gpu_kernel<<<grid, block>>>();

	CUDA_ERR_CHECK( hipGetLastError() );
	CUDA_ERR_CHECK( hipDeviceSynchronize() );
	return 0;
}
